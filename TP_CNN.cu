#include "hip/hip_runtime.h"
// INCLUDE

#include <stdlib.h>
#include <stdio.h>



// PART 1 : Première couches du réseau de neurone LeNet-5 : Convolution 2D et subsampling

// Layer 1 - Génération des données de test

// Initialization of matrix function

void MatrixInit(float *M, int type, int k, int l, int m ) { 
    if (type == 0){
        for (int i = 0; i < k*l*m; i++){
            M[i] = 0;
        }
    }

    if (type == 1){
        for (int i = 0; i < k*l*m; i++){
            M[i] = 1;
        }
    }

    else if (type == 2){
        for (int i = 0; i < k*l*m; i++){
            M[i] = 1.0 -(2*float(rand())) / float(RAND_MAX);;
        }
    }
}



// Print of matrix function

void MatrixPrint(float *M, int n, int p) {
    for(int line = 0; line < n; line++){
        for(int row = line*p; row < p*(line+1); row++){
            printf("%f   ", M[row]);
        }
        printf("\n");
    }
    printf("\n");
}





// Layer 2 - Convolution

// 2D Convolution function

__global__ void cudaConv2D(float* M, float* kernel, float* Mount, int M_line, int M_row, int kernel_size, int Mount_line, int Mount_row, int nb_kernel){
   
    int line = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp_sum;

    if (line < Mount_line && row < Mount_row){ 

        int tot_kernel = kernel_size * kernel_size;
        int tot_Mount = Mount_line * Mount_row;

        for (int n_k = 0; n_k < nb_kernel; n_k++){
            tmp_sum = 0.0;
            for (int kernel_line = 0; kernel_line < kernel_size; kernel_line++) {
                for (int kernel_row = 0; kernel_row < kernel_size; kernel_row++) {
                    tmp_sum += M[(line + kernel_line) * M_row + (row + kernel_row)] * kernel[kernel_row * kernel_size + kernel_row + n_k * tot_kernel];  
                }
            }
            Mount[line * Mount_row + row + n_k * tot_Mount] = tmp_sum;
        }
    }
}





// Layer 3 - 2D Subsampling

// 2D Subsampling function

__global__ void cudaSubsampling(float* M, float* Mount, int M_line, int M_row, int M_prof, int meanpool_size, int Mount_line, int Mount_row){

    int lig = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (lig % meanpool_size == 0 && col % meanpool_size == 0){ 

        float tmp_sum;
        int tot_meanpool = meanpool_size * meanpool_size;
        int tot_M = M_ligne * M_row;
        int tot_Mout = Mount_line * Mount_row;

        for (int n_prof = 0; n_prof < M_prof; n_prof++){
            tmp_sum = 0.0;
            for (int meanpool_line = 0; meanpool_line < meanpool_size; meanpool_line++) {
                for (int meanpool_col = 0; meanpool_row < meanpool_size; meanpool_row++) {
                    tmp_sum += M[(lig + meanpool_line) * M_row + row + meanpool_row + n_prof * tot_M] / tot_meanpool;
                }
            }

            if (line == 0){
                Mount[line * Mount_row + (row / meanpool_size) + n_prof * tot_Mount] = tmp_sum;
            }

            else if (row == 0){
                Mount[(line / meanpool_size) * Mount_row + row + n_prof * tot_Mount] = tmp_sum;
            }

            else{
                Mount[(line / meanpool_size) * Mount_row + (row / meanpool_size) + n_prof * tot_Mount] = tmp_sum;
            }
        }
    }
}


// Activation 

__device__ float* activation_tanh(float* M, int M_line, int M_row, int M_prof){

    int line = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (line < M_line && row < M_row){
        int tot_M = M_line * M_row;
        for (int n_prof = 0; n_prof < M_prof; n_prof++){
            M[line * M_row + col + n_prof * tot_M] = tanh(M[line * M_row + row + n_prof * tot_M]);
        }
    }
    return M;
}



__global__ void cudaTanh(float* M, int M_ligne, int M_colonne, int M_prof){
    activation_tanh(M, M_ligne, M_colonne, M_prof);
}






// MAIN

int main(int argc, char*argv[]) {
    
    // LAYER 1
    
    // Initialization
    float *raw_data, *C1_data, *S1_data, *C1_kernel;
    float *cuda_raw_data, *cuda_C1_data, *cuda_S1_data, *cuda_C1_kernel;

    // Memory allocation for the CPU
    raw_data = (float*)malloc(1*32*32*sizeof(float));
    C1_data = (float*)malloc(6*28*28*sizeof(float));
    S1_data = (float*)malloc(6*14*14*sizeof(float));
    C1_kernel = (float*)malloc(6*5*5*sizeof(float));
    
    // Matrix initialization
    MatrixInit(raw_data, 2, 1, 32, 32);  
    MatrixInit(C1_data, 0, 6, 28, 28);
    MatrixInit(S1_data, 0, 6, 14, 14);  
    MatrixInit(C1_kernel, 2, 6, 5, 5);

    // Memory allocation for the GPU
    hipMalloc((void**)&cuda_raw_data, sizeof(float)*1*32*32);
    hipMalloc((void**)&cuda_C1_data, sizeof(float)*6*28*28);
    hipMalloc((void**)&cuda_S1_data, sizeof(float)*6*14*14);
    hipMalloc((void**)&cuda_C1_kernel, sizeof(float)*6*5*5);

    // Copy of the data from the CPU to the GPU
    hipMemcpy(cuda_raw_data, raw_data, sizeof(float)*1*32*32, hipMemcpyHostToDevice); 
    hipMemcpy(cuda_C1_data, C1_data, sizeof(float)*6*28*28, hipMemcpyHostToDevice);
    hipMemcpy(cuda_S1_data, S1_data, sizeof(float)*6*14*14, hipMemcpyHostToDevice); 
    hipMemcpy(cuda_C1_kernel, C1_kernel, sizeof(float)*6*5*5, hipMemcpyHostToDevice);



    // LAYER 2

printf("Initial matrix :\n\n");
MatrixPrint(raw_data, 32, 32);
printf("\n\n");

printf("Convolution Kernel :\n\n");
MatrixPrint(C1_kernel, 5, 5);
printf("\n\n");

    // 2D Convolution
    dim3 block_dim(32,32);
    dim3 grid_dim(1,1);
    printf("2D Convolution result :\n\n");
    cudaConv2D<<<block_dim, grid_dim>>>(cuda_raw_data, cuda_C1_kernel, cuda_C1_data, 32, 32, 5, 6, 28, 28);
    hipDeviceSynchronize();
    hipMemcpy(C1_data, cuda_C1_data, sizeof(float)*6*28*28, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    MatrixPrint(C1_data, 28, 28);
}



//      nvcc TP_CNN.cu -o TP_CNN_cuda
//      ./TP_CNN.cuda
